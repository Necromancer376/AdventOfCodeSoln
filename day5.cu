#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <limits>
#include <hip/hip_runtime.h>

__device__ long getVal(const long* vec, long x, long size) {
    for (long i = 0; i < size; i += 3) {
        if (x >= vec[i + 1] && x < vec[i + 1] + vec[i + 2]) {
            return vec[i] + (x - vec[i + 1]);
        }
    }
    return x;
}

__global__ void computeKernel(const long* seeds, long n, const long* seed_soil, const long* soil_fert,
                              const long* fert_water, const long* water_light, const long* light_temp,
                              const long* temp_humd, const long* humd_loc, long* result) {
    long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        long ans = LONG_MAX;

        for (long j = 0; j < seeds[tid + 1]; j++) {
            long soil = getVal(seed_soil, seeds[tid] + j, 3 * seed_soil[0]);
            long fert = getVal(soil_fert, soil, 3 * soil_fert[0]);
            long water = getVal(fert_water, fert, 3 * fert_water[0]);
            long light = getVal(water_light, water, 3 * water_light[0]);
            long temp = getVal(light_temp, light, 3 * light_temp[0]);
            long humd = getVal(temp_humd, temp, 3 * temp_humd[0]);
            long loc = getVal(humd_loc, humd, 3 * humd_loc[0]);

            atomicMin(&ans, loc);
        }

        result[tid] = ans;
    }
}

int main() {
    long n;
    std::cin >> n;

    // Allocate and copy data to the device
    long *d_seeds, *d_seed_soil, *d_soil_fert, *d_fert_water, *d_water_light, *d_light_temp, *d_temp_humd, *d_humd_loc, *d_result;
    hipMalloc((void**)&d_seeds, n * sizeof(long));
    hipMalloc((void**)&d_seed_soil, 3 * sizeof(long) * seed_soil.size());
    hipMalloc((void**)&d_soil_fert, 3 * sizeof(long) * soil_fert.size());
    hipMalloc((void**)&d_fert_water, 3 * sizeof(long) * fert_water.size());
    hipMalloc((void**)&d_water_light, 3 * sizeof(long) * water_light.size());
    hipMalloc((void**)&d_light_temp, 3 * sizeof(long) * light_temp.size());
    hipMalloc((void**)&d_temp_humd, 3 * sizeof(long) * temp_humd.size());
    hipMalloc((void**)&d_humd_loc, 3 * sizeof(long) * humd_loc.size());
    hipMalloc((void**)&d_result, n * sizeof(long));

    hipMemcpy(d_seeds, seeds.data(), n * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(d_seed_soil, seed_soil.data(), 3 * sizeof(long) * seed_soil.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_soil_fert, soil_fert.data(), 3 * sizeof(long) * soil_fert.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_fert_water, fert_water.data(), 3 * sizeof(long) * fert_water.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_water_light, water_light.data(), 3 * sizeof(long) * water_light.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_light_temp, light_temp.data(), 3 * sizeof(long) * light_temp.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_temp_humd, temp_humd.data(), 3 * sizeof(long) * temp_humd.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_humd_loc, humd_loc.data(), 3 * sizeof(long) * humd_loc.size(), hipMemcpyHostToDevice);

    // Set up the grid and block dimensions
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    computeKernel<<<gridSize, blockSize>>>(d_seeds, n, d_seed_soil, d_soil_fert, d_fert_water, d_water_light,
                                            d_light_temp, d_temp_humd, d_humd_loc, d_result);

    // Copy the result back to the host
    long* h_result = new long[n];
    hipMemcpy(h_result, d_result, n * sizeof(long), hipMemcpyDeviceToHost);

    // Find the minimum value from the results
    long ans = LONG_MAX;
    for (long i = 0; i < n; i++) {
        ans = std::min(ans, h_result[i]);
    }

    // 'ans' now contains the minimum value across all iterations
    std::cout << "Minimum value: " << ans << std::endl;

    // Clean up
    delete[] h_result;
    hipFree(d_seeds);
    hipFree(d_seed_soil);
    hipFree(d_soil_fert);
    hipFree(d_fert_water);
    hipFree(d_water_light);
    hipFree(d_light_temp);
    hipFree(d_temp_humd);
    hipFree(d_humd_loc);
    hipFree(d_result);

    return 0;
}
